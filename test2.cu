#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include "hip/hip_runtime.h"
#define N 16

__global__ void MatAdd(float A[N][N], float B[N][N], float C[N][N])
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;
if (i < N && j < N)
C[i][j] = A[i][j] + B[i][j];
}

int main()
{
float A[N][N];
float B[N][N];
float C[N][N];

float (*d_A)[N]; //pointers to arrays of dimension N
float (*d_B)[N];
float (*d_C)[N];

for(int i = 0; i < N; i++) {
    for(int j = 0; j < N; j++) {
        A[i][j] = i;
        B[i][j] = j;
    }
}       

//allocation
hipMalloc((void**)&d_A, (N*N)*sizeof(float));
hipMalloc((void**)&d_B, (N*N)*sizeof(float));
hipMalloc((void**)&d_C, (N*N)*sizeof(float));

//copying from host to device
hipMemcpy(d_A, A, (N*N)*sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(d_B, B, (N*N)*sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(d_C, C, (N*N)*sizeof(float), hipMemcpyHostToDevice);

// Kernel invocation
dim3 threadsPerBlock(16, 16);
dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);
MatAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);

//copying from device to host
hipMemcpy(A, (d_A), (N*N)*sizeof(float), hipMemcpyDeviceToHost);
hipMemcpy(B, (d_B), (N*N)*sizeof(float), hipMemcpyDeviceToHost);
hipMemcpy(C, (d_C), (N*N)*sizeof(float), hipMemcpyDeviceToHost);


for(int i=0;i<N;i++)
      for(int j=0;j<N;j++)
      {
        printf("%d\n",C[i][j]);

      }

}
